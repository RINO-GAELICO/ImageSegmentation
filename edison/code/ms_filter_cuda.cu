#include "hip/hip_runtime.h"
#include "msImageProcessor.h"

#include <iostream>
#include <hip/hip_runtime.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

__global__ void msImageProcessorKernel(double *d_sdata, int *d_buckets, double *d_weightMap, float *d_msRawData,
                                       int *d_slist, double hiLTr, int nBuck1, int nBuck2,
                                       int nBuck3, int width, int height, int *d_bucNeigh,
                                       double sMins, double sigmaS, double sigmaR)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    // printf("i = %d\n", i);
    int L = width * height;
    int N = 1;
    int lN = N + 2;

    if (i < L)
    {
        double yk[3];
        double Mh[3];

        int idxs = i * lN;

        for (int j = 0; j < lN; j++)
        {
            yk[j] = d_sdata[idxs + j];
        }

        // Initialize mean shift vector with 0,0,0
        for (int j = 0; j < lN; j++)
        {
            Mh[j] = 0;
        }

        double wsuml = 0;
        int cBuck1 = static_cast<int>(yk[0]) + 1;
        int cBuck2 = static_cast<int>(yk[1]) + 1;
        int cBuck3 = static_cast<int>(yk[2] - sMins) + 1;

        int cBuck = cBuck1 + nBuck1 * (cBuck2 + nBuck2 * cBuck3);

        int neighborCount = 27;

        for (int j = 0; j < neighborCount; j++)
        {
            int idxd = d_buckets[cBuck + d_bucNeigh[j]];

            // printf("idxd = %d\n", idxd);
            while (idxd >= 0)
            {
                idxs = lN * idxd;

                double el = d_sdata[idxs] - yk[0];
                double diff = el * el;
                el = d_sdata[idxs + 1] - yk[1];
                diff += el * el;

                if (diff < 1.0)
                {
                    el = d_sdata[idxs + 2] - yk[2];
                    if (yk[2] > hiLTr)
                        diff = 4 * el * el;
                    else
                        diff = el * el;

                    if (diff < 1.0)
                    {
                        double weight = 1 - d_weightMap[idxd];
                        for (int k = 0; k < lN; k++)
                            Mh[k] += weight * d_sdata[idxs + k];
                        wsuml += weight;
                    }
                }
                idxd = d_slist[idxd];
            }
        }

        if (wsuml > 0)
        {
            for (int j = 0; j < lN; j++)
            {
                Mh[j] = Mh[j] / wsuml - yk[j];
            }
        }
        else
        {
            for (int j = 0; j < lN; j++)
            {
                Mh[j] = 0;
            }
        }

        double mvAbs = ((Mh[0] * Mh[0]) + (Mh[1] * Mh[1])) * sigmaS * sigmaS + (Mh[2] * Mh[2]) * sigmaR * sigmaR;

        int iterationCount = 1;
        while ((mvAbs >= EPSILON) && (iterationCount < LIMIT))
        {
            for (int j = 0; j < lN; j++)
                yk[j] += Mh[j];

            for (int j = 0; j < lN; j++)
                Mh[j] = 0;
            wsuml = 0;

            cBuck1 = static_cast<int>(yk[0]) + 1;
            cBuck2 = static_cast<int>(yk[1]) + 1;
            cBuck3 = static_cast<int>(yk[2] - sMins) + 1;
            cBuck = cBuck1 + nBuck1 * (cBuck2 + nBuck2 * cBuck3);

            for (int j = 0; j < neighborCount; j++)
            {
                int idxd = d_buckets[cBuck + j];

                while (idxd >= 0)
                {
                    idxs = lN * idxd;

                    double el = d_sdata[idxs] - yk[0];
                    double diff = el * el;
                    el = d_sdata[idxs + 1] - yk[1];
                    diff += el * el;

                    if (diff < 1.0)
                    {
                        el = d_sdata[idxs + 2] - yk[2];
                        if (yk[2] > hiLTr)
                            diff = 4 * el * el;
                        else
                            diff = el * el;

                        if (diff < 1.0)
                        {
                            double weight = 1 - d_weightMap[idxd];
                            for (int k = 0; k < lN; k++)
                                Mh[k] += weight * d_sdata[idxs + k];
                            wsuml += weight;
                        }
                    }
                    idxd = d_slist[idxd];
                }
            }

            if (wsuml > 0)
            {
                for (int j = 0; j < lN; j++)
                    Mh[j] = Mh[j] / wsuml - yk[j];
            }
            else
            {
                for (int j = 0; j < lN; j++)
                    Mh[j] = 0;
            }

            mvAbs = ((Mh[0] * Mh[0]) + (Mh[1] * Mh[1])) * sigmaS * sigmaS + (Mh[2] * Mh[2]) * sigmaR * sigmaR;

            iterationCount++;
        }

        for (int j = 0; j < lN; j++)
        {
            yk[j] += Mh[j];
        }

        d_msRawData[i] = static_cast<float>(yk[2] * sigmaR);
        return;
    }
}

void msImageProcessor::Filter_cuda(float sigmaS, float sigmaR)
{
    // Host code

    // Allocate memory on the device
    // define lN
    int lN = N + 2;

    // let's use some temporary data
    double *sdata;
    sdata = new double[lN * L];
    // index the data in the 3d buckets (x, y, L)
    int *buckets;
    int *slist;
    slist = new int[L];

    // copy the scaled data
    int idxs, idxd;
    idxs = idxd = 0;

    std::cout << "This is just a test" << std::endl;
    // WE FOCUS ON GRAYSCALE
    if (N == 1)
    {
        for (int i = 0; i < L; i++)
        {
            sdata[idxs++] = (i % width) / sigmaS;
            sdata[idxs++] = (i / width) / sigmaS;
            sdata[idxs++] = data[idxd++] / sigmaR;
        }
    }

    int bucNeigh[27]; // 27 because it is 3x3x3

    double sMins;    // just for L
    double sMaxs[3]; // for all

    // we store the max values of each dimension
    //  the range of the scaled values for the intensity
    sMaxs[0] = width / sigmaS;
    sMaxs[1] = height / sigmaS;
    sMins = sMaxs[2] = sdata[2];
    idxs = 2;
    double cval;
    // find the min and max values of the intensity
    for (int i = 0; i < L; i++)
    {
        cval = sdata[idxs];
        if (cval < sMins)
            sMins = cval;
        else if (cval > sMaxs[2])
            sMaxs[2] = cval;

        idxs += lN;
    }

    int nBuck1, nBuck2, nBuck3;
    int cBuck1, cBuck2, cBuck3, cBuck;
    nBuck1 = (int)(sMaxs[0] + 3);
    nBuck2 = (int)(sMaxs[1] + 3);
    nBuck3 = (int)(sMaxs[2] - sMins + 3);
    buckets = new int[nBuck1 * nBuck2 * nBuck3];
    for (int i = 0; i < (nBuck1 * nBuck2 * nBuck3); i++)
    {
        buckets[i] = -1;
    }

    idxs = 0;
    for (int i = 0; i < L; i++)
    {
        // find bucket for current data and add it to the list
        cBuck1 = (int)sdata[idxs] + 1;
        cBuck2 = (int)sdata[idxs + 1] + 1;
        cBuck3 = (int)(sdata[idxs + 2] - sMins) + 1;
        cBuck = cBuck1 + nBuck1 * (cBuck2 + nBuck2 * cBuck3);

        slist[i] = buckets[cBuck];
        buckets[cBuck] = i;

        idxs += lN;
    }

    idxd = 0;
    for (cBuck1 = -1; cBuck1 <= 1; cBuck1++)
    {
        for (cBuck2 = -1; cBuck2 <= 1; cBuck2++)
        {
            for (cBuck3 = -1; cBuck3 <= 1; cBuck3++)
            {
                bucNeigh[idxd++] = cBuck1 + nBuck1 * (cBuck2 + nBuck2 * cBuck3);
            }
        }
    }
    
    double hiLTr = 80.0 / sigmaR;

    // Allocate memory on the device
    double *d_sdata;
    int *d_buckets;
    double *d_weightMap;
    int *d_slist;
    int *d_bucNeigh;
    float *d_msRawData;

    gpuErrchk(hipMalloc(&d_sdata, sizeof(double) * lN * L));
    gpuErrchk(hipMalloc(&d_buckets, sizeof(int) * nBuck1 * nBuck2 * nBuck3));
    gpuErrchk(hipMalloc(&d_weightMap, sizeof(double) * L));
    gpuErrchk(hipMalloc(&d_slist, sizeof(int) * L));
    gpuErrchk(hipMalloc(&d_bucNeigh, sizeof(int) * 27));
    gpuErrchk(hipMalloc(&d_msRawData, sizeof(float) * L));

    // Transfer data from host to device
    hipMemcpy(d_sdata, sdata, sizeof(double) * lN * L, hipMemcpyHostToDevice);
    hipMemcpy(d_buckets, buckets, sizeof(int) * nBuck1 * nBuck2 * nBuck3, hipMemcpyHostToDevice);
    hipMemcpy(d_weightMap, weightMap, sizeof(double) * L, hipMemcpyHostToDevice);
    hipMemcpy(d_slist, slist, sizeof(int) * L, hipMemcpyHostToDevice);
    hipMemcpy(d_bucNeigh, bucNeigh, sizeof(int) * 27, hipMemcpyHostToDevice);

    // Launch the CUDA kernel

    const int threadsPerBlock = 1024;
    const int numberBlocks = (L + threadsPerBlock - 1) / threadsPerBlock;

    // let's measure the time

    std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();

    msImageProcessorKernel<<<numberBlocks, threadsPerBlock>>>(d_sdata, d_buckets, d_weightMap, d_msRawData, d_slist,
                                                    hiLTr, nBuck1, nBuck2, nBuck3, width, height, d_bucNeigh,
                                                    sMins, sigmaS, sigmaR);

    // Check for CUDA errors
    hipError_t hipError_t = hipGetLastError();
    if (hipError_t != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(hipError_t) << std::endl;
        // Handle the error appropriately
    }

    // Synchronize to ensure the kernel has completed
    hipDeviceSynchronize();

    std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;
    std::cout << "Total GPU processing time without copying:\t" << elapsed.count() << " s" << std::endl;

    // Copy the result back to the host
    hipMemcpy(msRawData, d_msRawData, sizeof(float) * L, hipMemcpyDeviceToHost);

    // Free allocated memory on the device
    hipFree(d_sdata);
    hipFree(d_buckets);
    hipFree(d_weightMap);
    hipFree(d_slist);
    hipFree(d_bucNeigh);
    hipFree(d_msRawData);
    return;
}
